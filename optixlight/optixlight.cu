#include "hip/hip_runtime.h"
#include <optix.h>
#include "optixlight.h"
#include <cuda/random.h>
#include <sutil/vec_math.h>


extern "C" {
__constant__ Params params;
}


static __forceinline__ __device__ void sample_sphere(const float u1, const float u2, float3& p)
{
    const float theta = 2.0f * M_PIf * u1;
    const float phi = acosf(2.0f * u2 - 1.0f);

    p.x = sinf(phi) * cosf(theta);
    p.y = sinf(phi) * sinf(theta);
    p.z = cosf(phi);
}


extern "C" __global__ void __raygen__rg()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned int seed = tea<4>(idx.x, params.seed);
    const float3 ray_origin = params.light_origin;
    float3 ray_direction;

    sample_sphere(rnd(seed), rnd(seed), ray_direction);

    // Trace the ray against our scene hierarchy
    unsigned int p0;
    optixTrace(
            params.handle,
            ray_origin,
            ray_direction,
            0.0f,                // Min intersection distance
            1e16f,               // Max intersection distance
            0.0f,                // rayTime -- used for motion blur
            OptixVisibilityMask(255), // Specify always visible
            OPTIX_RAY_FLAG_NONE,
            0,                   // SBT offset   -- See SBT discussion
            1,                   // SBT stride   -- See SBT discussion
            0,                   // missSBTIndex -- See SBT discussion
            p0);
    atomicAdd(&params.counts[p0], 1);
}

extern "C" __global__ void __miss__ms()
{
    optixSetPayload_0(0);
}


extern "C" __global__ void __closesthit__ch()
{
    HitData* rt_data  = reinterpret_cast<HitData*>( optixGetSbtDataPointer() );
    const float3 ray_dir = optixGetWorldRayDirection();
    const float3 poi = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_dir;
    const float4 poi4 = make_float4(poi.x, poi.y, poi.z, 1);
    const int s = static_cast<int>(dot(poi4, rt_data->m0));
    const int t = static_cast<int>(dot(poi4, rt_data->m1));

    optixSetPayload_0(1 + rt_data->idx);
}
