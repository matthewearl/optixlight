#include "hip/hip_runtime.h"
#include <optix.h>
#include "optixlight.h"
#include <cuda/random.h>
#include <sutil/vec_math.h>


extern "C" {
__constant__ Params params;
}


static __forceinline__ __device__ void sample_sphere(const float u1, const float u2, float3& p)
{
    const float theta = 2.0f * M_PIf * u1;
    const float phi = acosf(2.0f * u2 - 1.0f);

    p.x = sinf(phi) * cosf(theta);
    p.y = sinf(phi) * sinf(theta);
    p.z = cosf(phi);
}


extern "C" __global__ void __raygen__rg()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned int seed = tea<4>(idx.y*params.width + idx.x, params.seed);
    const float3 ray_origin = params.light_origin;
    float3 ray_direction;

    sample_sphere(rnd(seed), rnd(seed), ray_direction);

    // Trace the ray against our scene hierarchy
    float3 result = make_float3( 0 );
    unsigned int p0;
    optixTrace(
            params.handle,
            ray_origin,
            ray_direction,
            0.0f,                // Min intersection distance
            1e16f,               // Max intersection distance
            0.0f,                // rayTime -- used for motion blur
            OptixVisibilityMask(255), // Specify always visible
            OPTIX_RAY_FLAG_NONE,
            0,                   // SBT offset   -- See SBT discussion
            1,                   // SBT stride   -- See SBT discussion
            0,                   // missSBTIndex -- See SBT discussion
            p0);
    params.counts[p0] += 1;
}

extern "C" __global__ void __miss__ms()
{
    optixSetPayload_0(0);
}


extern "C" __global__ void __closesthit__ch()
{
    const int prim_idx = optixGetPrimitiveIndex();
    optixSetPayload_0(1 + prim_idx);
}
